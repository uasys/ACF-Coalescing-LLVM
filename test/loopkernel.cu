
#include <hip/hip_runtime.h>

__global__
void kernel(int *a, int *b) {
    if(threadIdx.x == 0) {
        a[threadIdx.x] = 0;
    }
    a[threadIdx.x] = b[threadIdx.x];
    a[threadIdx.x] = b[2*threadIdx.x];
    if(threadIdx.x%2 == 0) {
        a[threadIdx.x] = 0;
    }
}

int main()
{
    int a[10] = {2};
    int b[10] = {1};

    int *a_d;
    int *b_d;
    hipMalloc( &a_d, sizeof(a) );
    hipMalloc( &b_d, sizeof(b) );
    hipMemcpy(a_d, a, sizeof(a), hipMemcpyHostToDevice );
    hipMemcpy(b_d, b, sizeof(b), hipMemcpyHostToDevice );
    kernel<<<10,1>>>(a,b);
    hipMemcpy(a, a_d, sizeof(a), hipMemcpyDeviceToHost );
    hipFree(a_d);
    hipFree(b_d);
}
