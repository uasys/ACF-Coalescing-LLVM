
#include <hip/hip_runtime.h>

__global__
void kernel(int *a, int *b) {
    a[threadIdx.x] = b[threadIdx.x];
}

int main()
{
    int a[10] = {2};
    int b[10] = {1};

    int *a_d;
    int *b_d;
    hipMalloc( &a_d, sizeof(a) );
    hipMalloc( &b_d, sizeof(b) );
    hipMemcpy(a_d, a, sizeof(a), hipMemcpyHostToDevice );
    hipMemcpy(b_d, b, sizeof(b), hipMemcpyHostToDevice );
    kernel<<<10,1>>>(a,b);
    hipMemcpy(a, a_d, sizeof(a), hipMemcpyDeviceToHost );
    hipFree(a_d);
    hipFree(b_d);
}
