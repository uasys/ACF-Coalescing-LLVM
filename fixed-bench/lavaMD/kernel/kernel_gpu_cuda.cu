#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------200
//	plasmaKernel_gpu_2
//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------200

#define ASGN_OFF(tgt, src, off) \
{ \
    tgt.v = &src.v[off]; \
    tgt.x = &src.x[off]; \
    tgt.y = &src.y[off]; \
    tgt.z = &src.z[off]; \
}

#define DOT_ARR(A,ai,B,bi) ((A.x[ai])*(B.x[bi])+(A.y[ai])*(B.y[bi])+(A.z[ai])*(B.z[bi]))	// STABLE

__global__ void kernel_gpu_cuda(par_str d_par_gpu,
								dim_str d_dim_gpu,
								box_str* d_box_gpu,
								FOUR_ARR d_rv_gpu,
								fp* d_qv_gpu,
								FOUR_ARR d_fv_gpu)
{

	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180
	//	THREAD PARAMETERS
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180

	int bx = blockIdx.x;																// get current horizontal block index (0-n)
	int tx = threadIdx.x;															// get current horizontal thread index (0-n)
	// int ax = bx*NUMBER_THREADS+tx;
	// int wbx = bx;
	int wtx = tx;

	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180
	//	DO FOR THE NUMBER OF BOXES
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180

	if(bx<d_dim_gpu.number_boxes){
	// while(wbx<box_indexes_counter){

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	Extract input parameters
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		// parameters
		fp a2 = 2.0*d_par_gpu.alpha*d_par_gpu.alpha;

		// home box
		int first_i;
		FOUR_ARR rA;
		FOUR_ARR fA;
        FOUR_ARR rA_shared;

		__shared__ fp ra_shr_v[NUMBER_PAR_PER_BOX];
		__shared__ fp ra_shr_x[NUMBER_PAR_PER_BOX];
		__shared__ fp ra_shr_y[NUMBER_PAR_PER_BOX];
		__shared__ fp ra_shr_z[NUMBER_PAR_PER_BOX];
        rA_shared.v=ra_shr_v;
        rA_shared.x=ra_shr_x;
        rA_shared.y=ra_shr_y;
        rA_shared.z=ra_shr_z;

		// nei box
		int pointer;
		int k = 0;
		int first_j;
		FOUR_ARR rB;
		fp* qB;
		int j = 0;

		FOUR_ARR rB_shared;
		__shared__ fp rb_shr_v[NUMBER_PAR_PER_BOX];
		__shared__ fp rb_shr_x[NUMBER_PAR_PER_BOX];
		__shared__ fp rb_shr_y[NUMBER_PAR_PER_BOX];
		__shared__ fp rb_shr_z[NUMBER_PAR_PER_BOX];
        rB_shared.v=rb_shr_v;
        rB_shared.x=rb_shr_x;
        rB_shared.y=rb_shr_y;
        rB_shared.z=rb_shr_z;

		__shared__ double qB_shared[100];

		// common
		fp r2;
		fp u2;
		fp vij;
		fp fs;
		fp fxij;
		fp fyij;
		fp fzij;
		THREE_VECTOR d;

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	Home box
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		//----------------------------------------------------------------------------------------------------------------------------------140
		//	Setup parameters
		//----------------------------------------------------------------------------------------------------------------------------------140

		// home box - box parameters
		first_i = d_box_gpu[bx].offset;

		// home box - distance, force, charge and type parameters
        ASGN_OFF(rA, d_rv_gpu, first_i);
        ASGN_OFF(fA, d_fv_gpu, first_i);

		//----------------------------------------------------------------------------------------------------------------------------------140
		//	Copy to shared memory
		//----------------------------------------------------------------------------------------------------------------------------------140

		// home box - shared memory
		while(wtx<NUMBER_PAR_PER_BOX){
			rA_shared.v[wtx] = rA.v[wtx];
			rA_shared.x[wtx] = rA.x[wtx];
			rA_shared.y[wtx] = rA.y[wtx];
			rA_shared.z[wtx] = rA.z[wtx];
			wtx = wtx + NUMBER_THREADS;
		}
		wtx = tx;

		// synchronize threads  - not needed, but just to be safe
		__syncthreads();

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	nei box loop
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		// loop over neiing boxes of home box
		for (k=0; k<(1+d_box_gpu[bx].nn); k++){

			//----------------------------------------50
			//	nei box - get pointer to the right box
			//----------------------------------------50

			if(k==0){
				pointer = bx;													// set first box to be processed to home box
			}
			else{
				pointer = d_box_gpu[bx].nei[k-1].number;							// remaining boxes are nei boxes
			}

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Setup parameters
			//----------------------------------------------------------------------------------------------------------------------------------140

			// nei box - box parameters
			first_j = d_box_gpu[pointer].offset;

			// nei box - distance, (force), charge and (type) parameters
            ASGN_OFF(rB, d_rv_gpu, first_j);

			qB = &d_qv_gpu[first_j];

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Setup parameters
			//----------------------------------------------------------------------------------------------------------------------------------140

			// nei box - shared memory
			while(wtx<NUMBER_PAR_PER_BOX){
				rB_shared.v[wtx] = rB.v[wtx];
				rB_shared.x[wtx] = rB.x[wtx];
				rB_shared.y[wtx] = rB.y[wtx];
				rB_shared.z[wtx] = rB.z[wtx];

				qB_shared[wtx] = qB[wtx];
				wtx = wtx + NUMBER_THREADS;
			}
			wtx = tx;

			// synchronize threads because in next section each thread accesses data brought in by different threads here
			__syncthreads();

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Calculation
			//----------------------------------------------------------------------------------------------------------------------------------140

			// loop for the number of particles in the home box
			// for (int i=0; i<nTotal_i; i++){
			while(wtx<NUMBER_PAR_PER_BOX){

				// loop for the number of particles in the current nei box
				for (j=0; j<NUMBER_PAR_PER_BOX; j++){

					// r2 = rA[wtx].v + rB[j].v - DOT(rA[wtx],rB[j]);
					// u2 = a2*r2;
					// vij= exp(-u2);
					// fs = 2.*vij;

					// d.x = rA[wtx].x  - rB[j].x;
					// fxij=fs*d.x;
					// d.y = rA[wtx].y  - rB[j].y;
					// fyij=fs*d.y;
					// d.z = rA[wtx].z  - rB[j].z;
					// fzij=fs*d.z;

					// fA[wtx].v +=  qB[j]*vij;
					// fA[wtx].x +=  qB[j]*fxij;
					// fA[wtx].y +=  qB[j]*fyij;
					// fA[wtx].z +=  qB[j]*fzij;



					r2 = (rA_shared.v[wtx]) + (rB_shared.v[j])
                        - DOT_ARR(rA_shared,wtx,rB_shared,j);
					u2 = a2*r2;
					vij= exp(-u2);
					fs = 2*vij;

					d.x = (fp)rA_shared.x[wtx]  - (fp)rB_shared.x[j];
					fxij=fs*d.x;
					d.y = (fp)rA_shared.y[wtx]  - (fp)rB_shared.y[j];
					fyij=fs*d.y;
					d.z = (fp)rA_shared.z[wtx]  - (fp)rB_shared.z[j];
					fzij=fs*d.z;

					fA.v[wtx] +=  (double)((fp)qB_shared[j]*vij);
					fA.x[wtx] +=  (double)((fp)qB_shared[j]*fxij);
					fA.y[wtx] +=  (double)((fp)qB_shared[j]*fyij);
					fA.z[wtx] +=  (double)((fp)qB_shared[j]*fzij);

				}

				// increment work thread index
				wtx = wtx + NUMBER_THREADS;

			}

			// reset work index
			wtx = tx;

			// synchronize after finishing force contributions from current nei box not to cause conflicts when starting next box
			__syncthreads();

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Calculation END
			//----------------------------------------------------------------------------------------------------------------------------------140

		}

		// // increment work block index
		// wbx = wbx + NUMBER_BLOCKS;

		// // synchronize - because next iteration will overwrite current shared memory
		// __syncthreads();

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	nei box loop END
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

	}

}
